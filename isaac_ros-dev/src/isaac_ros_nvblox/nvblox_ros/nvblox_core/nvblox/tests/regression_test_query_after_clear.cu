#include "hip/hip_runtime.h"
/*
Copyright 2024 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <gtest/gtest.h>

#include "nvblox/core/types.h"
#include "nvblox/core/unified_ptr.h"
#include "nvblox/gpu_hash/gpu_layer_view.h"
#include "nvblox/gpu_hash/internal/cuda/gpu_hash_interface.cuh"
#include "nvblox/gpu_hash/internal/cuda/gpu_indexing.cuh"
#include "nvblox/map/common_names.h"
#include "nvblox/map/layer.h"

using namespace nvblox;

__global__ void copyBlockAtOrigin(
    Index3DDeviceHashMapType<EsdfBlock> block_hash, EsdfBlock* output_block) {
  const Index3D block_idx(blockIdx.x, blockIdx.y, blockIdx.z);
  const Index3D thread_idx(threadIdx.x, threadIdx.y, threadIdx.z);
  __shared__ EsdfBlock* block_ptr;
  if ((thread_idx.array() == Index3D::Zero().array()).all()) {
    block_ptr = getBlockPtr(block_hash, block_idx);
  }
  __syncthreads();

  if (block_ptr == nullptr) {
    return;
  }

  // Copy to output
  output_block->voxels[threadIdx.z][threadIdx.y][threadIdx.x]
      .squared_distance_vox =
      block_ptr->voxels[threadIdx.z][threadIdx.y][threadIdx.x]
          .squared_distance_vox;
}

float getTestValue(const Index3D& idx) {
  constexpr int kMaxValue = 1000;
  return static_cast<float>(Index3DHash()(idx) % kMaxValue);
}

void checkBlockValues(const EsdfBlock& block,
                      std::function<float(const Index3D&)> get_value) {
  for (int x = 0; x < EsdfBlock::kVoxelsPerSide; x++) {
    for (int y = 0; y < EsdfBlock::kVoxelsPerSide; y++) {
      for (int z = 0; z < EsdfBlock::kVoxelsPerSide; z++) {
        EXPECT_EQ(block.voxels[x][y][z].squared_distance_vox,
                  get_value(Index3D(x, y, z)));
      }
    }
  }
}

void setBlockValues(EsdfBlock* block,
                    std::function<float(const Index3D&)> get_value) {
  for (int x = 0; x < EsdfBlock::kVoxelsPerSide; x++) {
    for (int y = 0; y < EsdfBlock::kVoxelsPerSide; y++) {
      for (int z = 0; z < EsdfBlock::kVoxelsPerSide; z++) {
        block->voxels[x][y][z].squared_distance_vox =
            get_value(Index3D(x, y, z));
      }
    }
  }
}

TEST(QueryAfterClear, SingleBlock) {
  constexpr float kVoxelSize = 0.05f;
  EsdfLayer esdf_layer(kVoxelSize, MemoryType::kUnified);
  auto block_ptr = esdf_layer.allocateBlockAtIndex(Index3D(0, 0, 0));

  auto get_test_value = [](const Index3D& idx) -> float {
    return getTestValue(idx);
  };
  auto get_zero = [](const Index3D&) -> float { return 0.0f; };

  // Test values
  setBlockValues(block_ptr.get(), get_test_value);

  // Output block
  auto output_block_ptr = EsdfBlock::allocate(MemoryType::kUnified);

  // Query on the GPU
  constexpr int kNumBlocks = 1;
  constexpr int kVoxelsPerSide = VoxelBlock<EsdfVoxel>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  copyBlockAtOrigin<<<kNumBlocks, kThreadsPerBlock>>>(
      esdf_layer.getGpuLayerView().getHash().impl_, output_block_ptr.get());
  hipDeviceSynchronize();
  hipPeekAtLastError();

  // Check
  LOG(INFO) << "Testing copy had an effect.";
  checkBlockValues(*output_block_ptr, get_test_value);

  // Clear the layer
  esdf_layer.clear();

  // Set the output block zero
  setBlockValues(output_block_ptr.get(), get_zero);

  // Try to copy again, should be a no-op.
  copyBlockAtOrigin<<<kNumBlocks, kThreadsPerBlock>>>(
      esdf_layer.getGpuLayerView().getHash().impl_, output_block_ptr.get());
  hipDeviceSynchronize();
  hipPeekAtLastError();

  // Check that the last kernel didn't affect the output.
  LOG(INFO) << "Testing copy didn't happen.";
  checkBlockValues(*output_block_ptr, get_zero);
}

int main(int argc, char** argv) {
  google::InitGoogleLogging(argv[0]);
  FLAGS_alsologtostderr = true;
  google::InstallFailureSignalHandler();
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
