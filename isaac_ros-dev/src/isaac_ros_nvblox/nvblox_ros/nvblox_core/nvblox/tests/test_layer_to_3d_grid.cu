/*
Copyright 2024 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <gtest/gtest.h>

#include "nvblox/core/hip_stream.h"
#include "nvblox/core/indexing.h"
#include "nvblox/geometry/bounding_boxes.h"
#include "nvblox/map/common_names.h"
#include "nvblox/map/internal/cuda/layer_to_3d_grid.cuh"
#include "nvblox/map/layer.h"
#include "nvblox/map/unified_3d_grid.h"

using namespace nvblox;

class LayerToUnifiedGridTest : public ::testing::Test {
 protected:
  LayerToUnifiedGridTest()
      : esdf_layer_(voxel_size_m_, MemoryType::kUnified),
        cuda_stream_(std::make_shared<CudaStreamOwning>()) {
    default_voxel_.squared_distance_vox = 1.0;
  }
  // Layer to test with
  const float voxel_size_m_ = 0.05;
  EsdfLayer esdf_layer_;

  // Default value of voxels in the dense
  EsdfVoxel default_voxel_;

  std::shared_ptr<CudaStreamOwning> cuda_stream_;
};

Index3D toGlobalVoxelIndex(const Index3D& block_idx, const Index3D& voxel_idx) {
  return block_idx * VoxelBlock<bool>::kVoxelsPerSide + voxel_idx;
}

float globalVoxelIndexToTestValue(const Index3D& global_voxel_idx) {
  Index3DHash hash;
  constexpr size_t kMaxValue = 1000;
  return static_cast<float>(hash(global_voxel_idx) % kMaxValue);
}

void setLayerToTestValues(EsdfLayer* layer_ptr) {
  callFunctionOnAllVoxels<EsdfVoxel>(
      layer_ptr, [](const Index3D& block_index, const Index3D& voxel_index,
                    EsdfVoxel* voxel) {
        voxel->squared_distance_vox = globalVoxelIndexToTestValue(
            toGlobalVoxelIndex(block_index, voxel_index));
      });
}

template <typename CellType, typename VoxelType>
void checkGridHasTestValue(
    const Unified3DGrid<CellType>& grid,
    const VoxelBlockLayer<VoxelType>& layer, const float default_value,
    std::function<float(const CellType& cell)> extract_test_value_op) {
  // Lambda to check if a global voxel index is in the layer.
  auto is_in_layer = [](const Index3D& global_voxel_idx,
                        const EsdfLayer& esdf_layer) -> bool {
    const Vector3f p_L =
        global_voxel_idx.cast<float>() * esdf_layer.voxel_size();
    const Index3D block_idx =
        getBlockIndexFromPositionInLayer(esdf_layer.block_size(), p_L);
    return esdf_layer.isBlockAllocated(block_idx);
  };
  // Iterate and check
  // - Values in the layer have been copied over.
  // - Blocks not in the layer are set to their default value.
  int num_checked = 0;
  int num_default = 0;
  const Index3D max_index = grid.min_index() + grid.aabb_size();
  for (int x = grid.min_index().x(); x < max_index.x(); x++) {
    for (int y = grid.min_index().y(); y < max_index.y(); y++) {
      for (int z = grid.min_index().z(); z < max_index.z(); z++) {
        const Index3D global_voxel_idx(x, y, z);
        EXPECT_TRUE(grid.isInsideGrid(global_voxel_idx));
        const float value = extract_test_value_op(grid(global_voxel_idx));
        constexpr float kEps = 1e-6;
        if (is_in_layer(global_voxel_idx, layer)) {
          EXPECT_NEAR(value, globalVoxelIndexToTestValue(global_voxel_idx),
                      kEps);
          num_checked++;
        } else {
          EXPECT_NEAR(value, default_value, kEps);
          num_default++;
        }
      }
    }
  }
  EXPECT_GT(num_checked, 0);
  LOG(INFO) << "num_checked: " << num_checked;
  LOG(INFO) << "num_default: " << num_default;
}

void checkGridHasTestValue(const Unified3DGrid<EsdfVoxel>& grid,
                           const EsdfLayer& layer,
                           const EsdfVoxel& default_voxel) {
  std::function<float(const EsdfVoxel& cell)> extract_test_value_op =
      [](const EsdfVoxel& voxel) -> float {
    return voxel.squared_distance_vox;
  };
  checkGridHasTestValue(grid, layer, default_voxel.squared_distance_vox,
                        extract_test_value_op);
}

TEST_F(LayerToUnifiedGridTest, WholeLayer) {
  // Allocate blocks
  const std::vector<Index3D> blocks = {
      Index3D(-1, 0, 0),  // NOLINT
      Index3D(0, 0, 0),   // NOLINT
      Index3D(1, 0, 0),   // NOLINT
      Index3D(1, 0, 1)    // NOLINT
  };
  esdf_layer_.allocateBlocksAtIndices(blocks, *cuda_stream_);

  // Set the test values
  setLayerToTestValues(&esdf_layer_);

  // AABB
  const auto aabb = getAABBOfAllocatedBlocks(esdf_layer_);

  // Convert to the grid.
  Unified3DGrid<EsdfVoxel> grid(MemoryType::kUnified);
  voxelLayerToDenseVoxelGridInAABBAsync(esdf_layer_, aabb, default_voxel_,
                                        &grid, *cuda_stream_);
  cuda_stream_->synchronize();
  hipPeekAtLastError();
  EXPECT_TRUE((aabb.sizes().array() > 0).all());
  EXPECT_TRUE((grid.aabb_size().array() > 0).all());

  // Check the output grid
  // Iterate over the output dense grid and check that everything has been
  // faithfully copied.
  checkGridHasTestValue(grid, esdf_layer_, default_voxel_);
}

TEST_F(LayerToUnifiedGridTest, PartialLayer) {
  // Allocate blocks
  const Index3D block_idx_1 = Index3D(0, 0, 0);
  const Index3D block_idx_2 = Index3D(1, 1, 1);
  const std::vector<Index3D> blocks = {
      block_idx_1,  // NOLINT
      block_idx_2,  // NOLINT
  };
  esdf_layer_.allocateBlocksAtIndices(blocks, *cuda_stream_);

  // Set the test values
  setLayerToTestValues(&esdf_layer_);

  // Construct an AABB which is in the middle of two blocks.
  const Index3D voxel_idx(3, 3, 3);  // voxle in the middle(ish)
  const Vector3f aabb_min = getCenterPositionFromBlockIndexAndVoxelIndex(
      esdf_layer_.block_size(), block_idx_1, voxel_idx);
  const Vector3f aabb_max = getCenterPositionFromBlockIndexAndVoxelIndex(
      esdf_layer_.block_size(), block_idx_2, voxel_idx);
  const auto aabb = AxisAlignedBoundingBox(aabb_min, aabb_max);

  // Convert to the grid.
  Unified3DGrid<EsdfVoxel> grid(MemoryType::kUnified);
  voxelLayerToDenseVoxelGridInAABBAsync(esdf_layer_, aabb, default_voxel_,
                                        &grid, *cuda_stream_);
  cuda_stream_->synchronize();
  hipPeekAtLastError();
  EXPECT_TRUE((aabb.sizes().array() > 0).all());
  EXPECT_TRUE((grid.aabb_size().array() > 0).all());

  // Check the output grid
  // Iterate over the output dense grid and check that everything has been
  // faithfully copied.
  checkGridHasTestValue(grid, esdf_layer_, default_voxel_);
}

TEST_F(LayerToUnifiedGridTest, CopyLayerToHost) {
  // Allocate blocks
  const Index3D block_idx = Index3D(0, 0, 0);
  esdf_layer_.allocateBlockAtIndex(block_idx);

  // Set the test values
  setLayerToTestValues(&esdf_layer_);

  // AABB
  const auto aabb = getAABBOfAllocatedBlocks(esdf_layer_);

  // Convert to the grid.
  Unified3DGrid<EsdfVoxel> grid_device(MemoryType::kDevice);
  voxelLayerToDenseVoxelGridInAABBAsync(esdf_layer_, aabb, default_voxel_,
                                        &grid_device, *cuda_stream_);
  EXPECT_TRUE((aabb.sizes().array() > 0).all());
  EXPECT_TRUE((grid_device.aabb_size().array() > 0).all());

  // Copy Grid to CPU
  Unified3DGrid<EsdfVoxel> grid_host(MemoryType::kHost);
  grid_host.copyFromAsync(grid_device, *cuda_stream_);
  cuda_stream_->synchronize();
  hipPeekAtLastError();

  // Check the output grid
  // Iterate over the output dense grid and check that everything has been
  // faithfully copied.
  checkGridHasTestValue(grid_host, esdf_layer_, default_voxel_);
}

struct ExtractDistanceFunctor {
  __device__ __inline__ float operator()(const EsdfVoxel& voxel) const {
    return voxel.squared_distance_vox;
  }
};

TEST_F(LayerToUnifiedGridTest, CopyLayerWithConversion) {
  // Allocate blocks
  const Index3D block_idx = Index3D(0, 0, 0);
  esdf_layer_.allocateBlockAtIndex(block_idx);

  // Set the test values
  setLayerToTestValues(&esdf_layer_);

  // AABB
  const auto aabb = getAABBOfAllocatedBlocks(esdf_layer_);

  // The conversion functor
  ExtractDistanceFunctor conversion_op;

  // The default output value
  const float default_value = -1.0f;

  // Convert to the grid.
  Unified3DGrid<float> grid(MemoryType::kUnified);
  voxelLayerToDenseVoxelGridInAABBAsync(esdf_layer_, aabb, default_value,
                                        conversion_op, &grid, *cuda_stream_);
  cuda_stream_->synchronize();
  hipPeekAtLastError();
  EXPECT_TRUE((aabb.sizes().array() > 0).all());
  EXPECT_TRUE((grid.aabb_size().array() > 0).all());

  // Check the output grid
  // Iterate over the output dense grid and check that everything has been
  // faithfully copied.
  checkGridHasTestValue<float, EsdfVoxel>(
      grid, esdf_layer_, default_value,
      [](const float& value) -> float { return value; });
}

int main(int argc, char** argv) {
  FLAGS_alsologtostderr = true;
  google::InitGoogleLogging(argv[0]);
  google::InstallFailureSignalHandler();
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
