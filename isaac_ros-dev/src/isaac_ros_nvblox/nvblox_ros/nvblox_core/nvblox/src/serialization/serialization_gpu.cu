#include "hip/hip_runtime.h"
/*
Copyright 2024 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "nvblox/serialization/internal/serialization_gpu.h"

#include <hip/hip_runtime.h>
#include <string>

#include "glog/logging.h"
#include "nvblox/core/internal/error_check.h"

namespace nvblox {

// Kernel that copies several vectors into a contigous chunk of memory
//
// Number of blocks:  Must equal num_vectors.
// Number of threads: Can be any positive value but a larger number than the
//   maximum vector size will not bring any gain.
//
// @param num_vectors        Number of vectors to serialize
// @param vectors            Vectors to serialize. Size: num_vectors
// @param offsets            Output buffer offsets. Last elements contain total
//                           num elements. Size: num_vectors+1
// @param serialized_buffer  Resulting buffer. Must have capacity for all
//                           elements
template <typename T>
void __global__ SerializeVectorsKernel(const int32_t num_vectors,
                                       const T** vectors,
                                       const int32_t* offsets,
                                       T* serialized_buffer) {
  const int32_t vector_index =
      blockIdx.x;  // Which vector does this block serialize?
  const int32_t element_index_start = threadIdx.x;

  // This should not happen if the kernel was launched with
  // num_blocks=num_vectors
  if (vector_index >= num_vectors) {
    return;
  }

  const int32_t offset = offsets[vector_index];
  const int32_t num_elements = offsets[vector_index + 1] - offset;

  // If vector size is larger than number of threads, we let each thread handle
  // several elements.
  for (int32_t index = element_index_start; index < num_elements;
       index += blockDim.x) {
    serialized_buffer[offset + index] = vectors[vector_index][index];
  }
}

template <typename LayerType, typename T>
void LayerSerializerGpuInternal<LayerType, T>::serializeAsync(
    const LayerType& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<T>& serialized_output, host_vector<int32_t>& offsets_output,
    std::function<
        std::pair<const T*, int>(const typename LayerType::BlockType* block)>
        get_data_and_size,
    const CudaStream hip_stream) {
  if (block_indices_to_serialize.empty()) {
    return;
  }

  // Iterate over all blocks to serialize, store their data pointers and
  // offsets
  offsets_output.resize(block_indices_to_serialize.size() + 1);
  vector_ptrs_.resize(block_indices_to_serialize.size());
  int32_t total_num_elements = 0;
  int32_t max_block_size = 0;
  for (size_t i = 0; i < block_indices_to_serialize.size(); ++i) {
    const typename LayerType::BlockType* block =
        layer.getBlockAtIndex(block_indices_to_serialize[i]).get();

    const auto data_and_size = get_data_and_size(block);
    offsets_output[i] = total_num_elements;
    vector_ptrs_[i] = data_and_size.first;
    total_num_elements += data_and_size.second;

    max_block_size = std::max<int32_t>(max_block_size, data_and_size.second);
  }

  // We'll need the total num of elements as well so we can compute the
  // size of the last vector
  offsets_output[offsets_output.size() - 1] = total_num_elements;

  // We use thread_id to determine which vector element to copy. This
  // allow for coalesced memory transfers since all threads in one warp
  // will read from contiguous memory.
  constexpr int32_t kMaxNumThreads = 1024;
  const int32_t num_threads = std::min(max_block_size, kMaxNumThreads);

  // Process one layer-block per cuda-block
  const int32_t num_cuda_blocks = block_indices_to_serialize.size();

  // Run serialization.
  serialized_output.resizeAsync(total_num_elements, hip_stream);
  if (num_threads > 0 && num_cuda_blocks > 0) {
    SerializeVectorsKernel<<<num_cuda_blocks, num_threads, 0, hip_stream>>>(
        block_indices_to_serialize.size(), vector_ptrs_.data(),
        offsets_output.data(), serialized_output.data());
  }

  checkCudaErrors(hipPeekAtLastError());
}

// Instantiation of serialize function for TSDF layer
template void LayerSerializerGpuInternal<TsdfLayer, TsdfVoxel>::serializeAsync(
    const TsdfLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<TsdfVoxel>& serialized_output,
    host_vector<int32_t>& offsets_output,
    std::function<std::pair<const TsdfVoxel*, int>(const TsdfBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

// Instantiation of serialize function for Color layer
template void
LayerSerializerGpuInternal<ColorLayer, ColorVoxel>::serializeAsync(
    const ColorLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<ColorVoxel>& serialized_output,
    host_vector<int32_t>& offsets_output,
    std::function<std::pair<const ColorVoxel*, int>(const ColorBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

// Instantiation of serialize function for Occupancy layer
template void
LayerSerializerGpuInternal<OccupancyLayer, OccupancyVoxel>::serializeAsync(
    const OccupancyLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<OccupancyVoxel>& serialized_output,
    host_vector<int32_t>& offsets_output,
    std::function<
        std::pair<const OccupancyVoxel*, int>(const OccupancyBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

// Instantiation of serialize function for Freespace layer
template void
LayerSerializerGpuInternal<FreespaceLayer, FreespaceVoxel>::serializeAsync(
    const FreespaceLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<FreespaceVoxel>& serialized_output,
    host_vector<int32_t>& offsets_output,
    std::function<
        std::pair<const FreespaceVoxel*, int>(const FreespaceBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

// Instantiation of serialize function for Esdf layer
template void LayerSerializerGpuInternal<EsdfLayer, EsdfVoxel>::serializeAsync(
    const EsdfLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<EsdfVoxel>& serialized_output,
    host_vector<int32_t>& offsets_output,
    std::function<std::pair<const EsdfVoxel*, int>(const EsdfBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

// Instantiation of serialize function for Mesh layer::Vector3f
template void LayerSerializerGpuInternal<MeshLayer, Vector3f>::serializeAsync(
    const MeshLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<Vector3f>& serialized_output,
    host_vector<int32_t>& offsets_output,
    std::function<std::pair<const Vector3f*, int>(const MeshBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

// Instantiation of serialize function for Mesh layer::Color
template void LayerSerializerGpuInternal<MeshLayer, Color>::serializeAsync(
    const MeshLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<Color>& serialized_output, host_vector<int32_t>& offsets_output,
    std::function<std::pair<const Color*, int>(const MeshBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

// Instantiation of serialize function for Mesh layer::float
template void LayerSerializerGpuInternal<MeshLayer, float>::serializeAsync(
    const MeshLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<float>& serialized_output, host_vector<int32_t>& offsets_output,
    std::function<std::pair<const float*, int>(const MeshBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

// Instantiation of serialize function for Mesh layer::int
template void LayerSerializerGpuInternal<MeshLayer, int>::serializeAsync(
    const MeshLayer& layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    host_vector<int>& serialized_output, host_vector<int32_t>& offsets_output,
    std::function<std::pair<const int*, int>(const MeshBlock* block)>
        get_data_and_size,
    const CudaStream hip_stream);

}  // namespace nvblox