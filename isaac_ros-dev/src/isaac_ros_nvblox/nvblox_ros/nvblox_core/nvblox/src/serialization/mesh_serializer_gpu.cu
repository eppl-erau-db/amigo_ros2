/*
Copyright 2023 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "nvblox/serialization/mesh_serializer_gpu.h"

#include <hip/hip_runtime.h>
#include <string>

#include "glog/logging.h"
#include "nvblox/core/internal/error_check.h"

namespace nvblox {

std::shared_ptr<const SerializedMesh> MeshSerializerGpu::serializeMesh(
    const MeshLayer& mesh_layer,
    const std::vector<Index3D>& block_indices_to_serialize,
    const CudaStream hip_stream) {
  vertex_serializer_.serializeAsync(
      mesh_layer, block_indices_to_serialize, serialized_mesh_->vertices,
      serialized_mesh_->vertex_block_offsets,
      [](const MeshBlock* mesh_block) -> const std::pair<const Vector3f*, int> {
        return std::make_pair(mesh_block->vertices.data(),
                              mesh_block->vertices.size());
      },
      hip_stream);

  color_serializer_.serializeAsync(
      mesh_layer, block_indices_to_serialize, serialized_mesh_->colors,
      serialized_mesh_->vertex_block_offsets,
      [](const MeshBlock* mesh_block) -> const std::pair<const Color*, int> {
        return std::make_pair(mesh_block->colors.data(),
                              mesh_block->colors.size());
      },
      hip_stream);

  triangle_index_serializer_.serializeAsync(
      mesh_layer, block_indices_to_serialize,
      serialized_mesh_->triangle_indices,
      serialized_mesh_->triangle_index_block_offsets,
      [](const MeshBlock* mesh_block) -> const std::pair<const int*, int> {
        return std::make_pair(mesh_block->triangles.data(),
                              mesh_block->triangles.size());
      },
      hip_stream);

  // Create an unique identifier for each block.
  serialized_mesh_->block_indices = block_indices_to_serialize;

  hip_stream.synchronize();

  return serialized_mesh_;
}

MeshSerializerGpu::MeshSerializerGpu()
    : serialized_mesh_(std::make_shared<SerializedMesh>()) {}

}  // namespace nvblox
